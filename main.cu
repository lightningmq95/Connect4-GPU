#include <iostream>
#include <vector>
#include <fstream>
#include <hip/hip_runtime.h>
#include <limits>
#include <climits>
#include <random>
#include <chrono>

#define ROWS 6
#define COLS 7

// prototype
__global__ void findBestMoveKernel(int* board, int player, int* scores, bool is_offensive);
__device__ int calculateBoardScore(int* board, int player);
void printBoard(int* board);
void logBoard(std::ofstream& logFile, int* board);
bool checkWin(int* board, int player);
bool isBoardFull(int* board);


// CUDA kernel to find the best move for a given player
__global__ void findBestMoveKernel(int* board, int player, int* scores, bool is_offensive) {
    int col = threadIdx.x;
    int opponent = (player == 1) ? 2 : 1;

    int temp_board[ROWS * COLS];
    for (int i = 0; i < ROWS * COLS; ++i) {
        temp_board[i] = board[i];
    }

    int row = -1;
    for (int r = ROWS - 1; r >= 0; --r) {
        if (temp_board[r * COLS + col] == 0) {
            row = r;
            break;
        }
    }

    if (row != -1) {
        // Simulate placing my piece
        temp_board[row * COLS + col] = player;
        int my_score = calculateBoardScore(temp_board, player);

        // Simulate opponent's best reply
        int opponent_best_reply_score = INT_MIN;
        for (int opp_col = 0; opp_col < COLS; ++opp_col) {
            int temp_board2[ROWS * COLS];
            for (int i = 0; i < ROWS * COLS; ++i) {
                temp_board2[i] = temp_board[i];
            }

            int opp_row = -1;
            for (int r = ROWS - 1; r >= 0; --r) {
                if (temp_board2[r * COLS + opp_col] == 0) {
                    opp_row = r;
                    break;
                }
            }

            if (opp_row != -1) {
                temp_board2[opp_row * COLS + opp_col] = opponent;
                int score = calculateBoardScore(temp_board2, opponent);
                if (score > opponent_best_reply_score) {
                    opponent_best_reply_score = score;
                }
            }
        }
        
        int final_score;
        if (is_offensive) {
            final_score = my_score * 2 - opponent_best_reply_score;
        } else {
            final_score = my_score - opponent_best_reply_score * 2;
        }
        scores[col] = final_score;

    } else {
        scores[col] = INT_MIN;
    }
}


// Device function to score a board state for a given player
__device__ int calculateBoardScore(int* board, int player) {
    int score = 0;
    int opponent = (player == 1) ? 2 : 1;

    // Score center column preference
    for (int r = 0; r < ROWS; ++r) {
        if (board[r * COLS + COLS / 2] == player) {
            score += 3;
        }
    }

    // Horizontal check
    for (int r = 0; r < ROWS; ++r) {
        for (int c = 0; c <= COLS - 4; ++c) {
            int p_count = 0, o_count = 0;
            for (int i = 0; i < 4; ++i) {
                if (board[r * COLS + c + i] == player) p_count++;
                else if (board[r * COLS + c + i] == opponent) o_count++;
            }
            if (p_count == 4) score += 10000;
            else if (p_count == 3 && o_count == 0) score += 100;
            else if (p_count == 2 && o_count == 0) score += 10;
        }
    }

    // Vertical check
    for (int c = 0; c < COLS; ++c) {
        for (int r = 0; r <= ROWS - 4; ++r) {
            int p_count = 0, o_count = 0;
            for (int i = 0; i < 4; ++i) {
                if (board[(r + i) * COLS + c] == player) p_count++;
                else if (board[(r + i) * COLS + c] == opponent) o_count++;
            }
            if (p_count == 4) score += 10000;
            else if (p_count == 3 && o_count == 0) score += 100;
            else if (p_count == 2 && o_count == 0) score += 10;
        }
    }

    // Positive diagonal check
    for (int r = 0; r <= ROWS - 4; ++r) {
        for (int c = 0; c <= COLS - 4; ++c) {
            int p_count = 0, o_count = 0;
            for (int i = 0; i < 4; ++i) {
                if (board[(r + i) * COLS + (c + i)] == player) p_count++;
                else if (board[(r + i) * COLS + (c + i)] == opponent) o_count++;
            }
            if (p_count == 4) score += 10000;
            else if (p_count == 3 && o_count == 0) score += 100;
            else if (p_count == 2 && o_count == 0) score += 10;
        }
    }

    // Negative diagonal check
    for (int r = 3; r < ROWS; ++r) {
        for (int c = 0; c <= COLS - 4; ++c) {
            int p_count = 0, o_count = 0;
            for (int i = 0; i < 4; ++i) {
                if (board[(r - i) * COLS + (c + i)] == player) p_count++;
                else if (board[(r - i) * COLS + (c + i)] == opponent) o_count++;
            }
            if (p_count == 4) score += 10000;
            else if (p_count == 3 && o_count == 0) score += 100;
            else if (p_count == 2 && o_count == 0) score += 10;
        }
    }
    return score;
}


void printBoard(int* board) {
    for (int r = 0; r < ROWS; ++r) {
        for (int c = 0; c < COLS; ++c) {
            char piece = '.';
            if (board[r * COLS + c] == 1) piece = 'X';
            else if (board[r * COLS + c] == 2) piece = 'O';
            std::cout << piece << " ";
        }
        std::cout << std::endl;
    }
    std::cout << "0 1 2 3 4 5 6" << std::endl;
}


void logBoard(std::ofstream& logFile, int* board) {
    for (int r = 0; r < ROWS; ++r) {
        for (int c = 0; c < COLS; ++c) {
            char piece = '.';
            if (board[r * COLS + c] == 1) piece = 'X';
            else if (board[r * COLS + c] == 2) piece = 'O';
            logFile << piece << " ";
        }
        logFile << std::endl;
    }
    logFile << "0 1 2 3 4 5 6" << std::endl;
    logFile << "-----------------" << std::endl;
}


bool checkWin(int* board, int player) {
    // Horizontal
    for (int r = 0; r < ROWS; ++r) {
        for (int c = 0; c <= COLS - 4; ++c) {
            if (board[r * COLS + c] == player && board[r * COLS + c + 1] == player && board[r * COLS + c + 2] == player && board[r * COLS + c + 3] == player) return true;
        }
    }
    // Vertical
    for (int c = 0; c < COLS; ++c) {
        for (int r = 0; r <= ROWS - 4; ++r) {
            if (board[r * COLS + c] == player && board[(r + 1) * COLS + c] == player && board[(r + 2) * COLS + c] == player && board[(r + 3) * COLS + c] == player) return true;
        }
    }
    // Positive diagonal
    for (int r = 0; r <= ROWS - 4; ++r) {
        for (int c = 0; c <= COLS - 4; ++c) {
            if (board[r * COLS + c] == player && board[(r + 1) * COLS + c + 1] == player && board[(r + 2) * COLS + c + 2] == player && board[(r + 3) * COLS + c + 3] == player) return true;
        }
    }
    // Negative diagonal
    for (int r = 3; r < ROWS; ++r) {
        for (int c = 0; c <= COLS - 4; ++c) {
            if (board[r * COLS + c] == player && board[(r - 1) * COLS + c + 1] == player && board[(r - 2) * COLS + c + 2] == player && board[(r - 3) * COLS + c + 3] == player) return true;
        }
    }
    return false;
}


bool isBoardFull(int* board) {
    for (int i = 0; i < COLS; ++i) {
        if (board[i] == 0) return false;
    }
    return true;
}

// game logic on the CPU
int main() {
    int* d_board;
    int* d_scores;
    char playAgain = 'y';

    // Allocate memory on the GPU once
    hipMalloc(&d_board, ROWS * COLS * sizeof(int));
    hipMalloc(&d_scores, COLS * sizeof(int));

    // --- NEW: Main loop to allow playing again ---
    while (playAgain == 'y' || playAgain == 'Y') {
        int h_board[ROWS * COLS] = {0}; // Reset host board for each new game

        // --- Setup for randomness ---
        unsigned seed = std::chrono::high_resolution_clock::now().time_since_epoch().count();
        std::mt19937 generator(seed);

        // --- Setup for logging (overwrites previous log) ---
        std::ofstream logFile("game_log.txt");
        if (!logFile.is_open()) {
            std::cerr << "Error: Could not open log file for writing." << std::endl;
            return 1;
        }
        logFile << "--- Connect 4 GPU Game Log ---" << std::endl;
        logBoard(logFile, h_board); // Log the initial empty board

        int turn = 0;
        // --- Inner game loop ---
        while (turn < ROWS * COLS) {
            int currentPlayer = (turn % 2) + 1;
            bool isOffensive = (currentPlayer == 1);

            std::cout << "\nTurn " << turn + 1 << ", Player " << (currentPlayer == 1 ? "X (Offensive)" : "O (Defensive)") << "'s move:" << std::endl;

            // Copy board from host to device
            hipMemcpy(d_board, h_board, ROWS * COLS * sizeof(int), hipMemcpyHostToDevice);

            // Launch kernel to find the best move
            findBestMoveKernel<<<1, COLS>>>(d_board, currentPlayer, d_scores, isOffensive);

            // Copy scores from device to host
            int h_scores[COLS];
            hipMemcpy(h_scores, d_scores, COLS * sizeof(int), hipMemcpyDeviceToHost);

            // Find the best move on the CPU
            int bestScore = std::numeric_limits<int>::min();
            for (int i = 0; i < COLS; ++i) {
                if (h_board[i] == 0) {
                    if (h_scores[i] > bestScore) {
                        bestScore = h_scores[i];
                    }
                }
            }

            // Randomly choose among the best moves
            std::vector<int> bestMoves;
            for (int i = 0; i < COLS; i++) {
                if (h_board[i] == 0 && h_scores[i] == bestScore) {
                    bestMoves.push_back(i);
                }
            }
            
            int bestMove = -1;
            if (!bestMoves.empty()) {
                std::uniform_int_distribution<int> distribution(0, bestMoves.size() - 1);
                bestMove = bestMoves[distribution(generator)];
            }

            // Make the move
            if (bestMove != -1) {
                for (int r = ROWS - 1; r >= 0; --r) {
                    if (h_board[r * COLS + bestMove] == 0) {
                        h_board[r * COLS + bestMove] = currentPlayer;
                        break;
                    }
                }
                logFile << "Turn " << turn + 1 << ": Player " << (currentPlayer == 1 ? 'X' : 'O') << " played in column " << bestMove << "." << std::endl;
                logBoard(logFile, h_board);
            }

            printBoard(h_board);

            // win check
            if (checkWin(h_board, currentPlayer)) {
                std::cout << "\nPlayer " << (currentPlayer == 1 ? "X (Offensive)" : "O (Defensive)") << " wins!" << std::endl;
                logFile << "\nPlayer " << (currentPlayer == 1 ? "X (Offensive)" : "O (Defensive)") << " wins!" << std::endl;
                break;
            }

            // draw check
            if (isBoardFull(h_board)) {
                std::cout << "\nIt's a draw!" << std::endl;
                logFile << "\nIt's a draw!" << std::endl;
                break;
            }

            turn++;
        }
        logFile.close();

        // play again
        std::cout << "\n--------------------------" << std::endl;
        std::cout << "Play again? (y/n): ";
        std::cin >> playAgain;
        std::cout << "--------------------------\n" << std::endl;
    }


    // Free GPU memory
    hipFree(d_board);
    hipFree(d_scores);
    
    std::cout << "Thanks for playing!" << std::endl;

    return 0;
}